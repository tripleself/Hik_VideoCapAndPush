#include "hip/hip_runtime.h"
﻿#include "PushStream.cuh"
#include <hip/hip_runtime.h>

// BGR到YUV420P的CUDA实现 - 优化版本
__global__ void bgr2yuv420p_kernel(const uint8_t *bgr, uint8_t *yuv_y, uint8_t *yuv_u, uint8_t *yuv_v,
                                   int width, int height, int bgr_step)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height)
    {
        int bgr_idx = y * bgr_step + x * 3;

        float b = bgr[bgr_idx];
        float g = bgr[bgr_idx + 1];
        float r = bgr[bgr_idx + 2];

        // ITU-R BT.601转换公式
        float Y = 0.299f * r + 0.587f * g + 0.114f * b;
        float U = -0.14713f * r - 0.28886f * g + 0.436f * b + 128.0f;
        float V = 0.615f * r - 0.51499f * g - 0.10001f * b + 128.0f;

        Y = fmaxf(0.0f, fminf(255.0f, Y));
        U = fmaxf(0.0f, fminf(255.0f, U));
        V = fmaxf(0.0f, fminf(255.0f, V));

        int y_idx = y * width + x;
        yuv_y[y_idx] = static_cast<uint8_t>(Y);

        if (x % 2 == 0 && y % 2 == 0)
        {
            int uv_idx = (y / 2) * (width / 2) + (x / 2);
            yuv_u[uv_idx] = static_cast<uint8_t>(U);
            yuv_v[uv_idx] = static_cast<uint8_t>(V);
        }
    }
}

void cudaBGR2YUV420P(const cv::Mat &bgr, uint8_t *yuv, int width, int height)
{
    if (bgr.empty() || bgr.type() != CV_8UC3 || !yuv)
        return;

    cv::Mat continuous_bgr = bgr.isContinuous() ? bgr : bgr.clone();

    uint8_t *d_bgr, *d_yuv_y, *d_yuv_u, *d_yuv_v;

    size_t bgrSize = continuous_bgr.total() * continuous_bgr.elemSize();
    size_t ySize = width * height;
    size_t uvSize = width * height / 4;
    int bgr_step = continuous_bgr.step[0];

    // 分配GPU内存
    if (hipMalloc(&d_bgr, bgrSize) != hipSuccess ||
        hipMalloc(&d_yuv_y, ySize) != hipSuccess ||
        hipMalloc(&d_yuv_u, uvSize) != hipSuccess ||
        hipMalloc(&d_yuv_v, uvSize) != hipSuccess)
    {
        hipFree(d_bgr);
        hipFree(d_yuv_y);
        hipFree(d_yuv_u);
        hipFree(d_yuv_v);
        return;
    }

    // 复制数据到GPU
    if (hipMemcpy(d_bgr, continuous_bgr.data, bgrSize, hipMemcpyHostToDevice) != hipSuccess)
    {
        hipFree(d_bgr);
        hipFree(d_yuv_y);
        hipFree(d_yuv_u);
        hipFree(d_yuv_v);
        return;
    }

    // 启动CUDA核函数
    dim3 block(16, 16);
    dim3 grid((width + 15) / 16, (height + 15) / 16);
    bgr2yuv420p_kernel<<<grid, block>>>(d_bgr, d_yuv_y, d_yuv_u, d_yuv_v, width, height, bgr_step);

    hipDeviceSynchronize();

    // 复制结果回CPU
    hipMemcpy(yuv, d_yuv_y, ySize, hipMemcpyDeviceToHost);
    hipMemcpy(yuv + ySize, d_yuv_u, uvSize, hipMemcpyDeviceToHost);
    hipMemcpy(yuv + ySize + uvSize, d_yuv_v, uvSize, hipMemcpyDeviceToHost);

    // 释放GPU内存
    hipFree(d_bgr);
    hipFree(d_yuv_y);
    hipFree(d_yuv_u);
    hipFree(d_yuv_v);
}